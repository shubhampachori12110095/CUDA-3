#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main (void) {
  int deviceCount = 0;
  hipError_t result = hipInit (0);
  
  result = hipGetDeviceCount (&deviceCount);
  for (int i=0; i<deviceCount; i++) {
    hipDevice_t device;
    char name[256];
    size_t bytes;
    int major, minor;
    CUdevprop prop;
    hipCtx_t ctx;
    size_t free, total;
    result = hipDeviceGet (&device, i);
    hipCtxCreate (&ctx, 0, device);
    printf ("device #           : %i\n"
            "=======================\n", i);
    hipDeviceGetName (name, 256, device);
    printf (" name              : %s\n", name);
    hipDeviceTotalMem (&bytes, device);
    printf (" memory            : %iMB\n", ((int) bytes)/(1024*1024));
    hipDeviceComputeCapability (&major, &minor, device);
    printf (" version           : %i.%i\n", major, minor);
    cuDeviceGetProperties (&prop, device);
    printf (" max threads/block : %i\n", prop.maxThreadsPerBlock);
    printf (" max thread size   : %ix%ix%i\n", 
            prop.maxThreadsDim[0],
	    prop.maxThreadsDim[1],
	    prop.maxThreadsDim[2]);
    printf (" max grid size     : %ix%ix%i\n", 
            prop.maxGridSize[0],
	    prop.maxGridSize[1],
	    prop.maxGridSize[2]);
    printf (" shared mem/block : %iKB\n", prop.sharedMemPerBlock/1024);
    printf (" constant memory  : %iKB\n", prop.totalConstantMemory/(1024));
    printf (" wrap size        : %i\n", prop.SIMDWidth);
    printf (" max mem pitch    : %i\n", prop.memPitch);
    printf (" registers/block  : %i\n", prop.regsPerBlock);
    printf (" clock rate       : %iMHz\n", prop.clockRate/1000);
    printf (" texture align    : %i\n", prop.textureAlign);
    printf ("\nMemory Info:\n");
    hipMemGetInfo (&free, &total);
    printf (" free             : %iMB\n", ((int) free)/(1024*1024));
    printf (" total            : %iMB\n", ((int) total)/(1024*1024));
    printf ("\n\n"); 
    cuCtxDetach(ctx);
  }
 
  return 0;
}
