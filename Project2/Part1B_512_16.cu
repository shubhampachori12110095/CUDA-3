
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#define N 512
#define TILE_SIZE 16
__global__ void MatrixMultiply(float *d_A, float *d_B, float *d_C, int length)
{
	
	/*int ty = threadIdx.y;*/
	/*int tx = threadIdx.x;*/
	int numcol = blockIdx.x*TILE_SIZE + threadIdx.x;
 	int numrow = blockIdx.y*TILE_SIZE + threadIdx.y;
 	float  cvalue = 0;
 	for (int k =0; k<N;++k){
        float d_Aelement =  d_A[numrow*N+k];
        float d_Belement =  d_B[k*N+numcol];
        cvalue += d_Aelement * d_Belement;
        }
d_C[numrow*N+numcol] = cvalue;
}
int main(){

	int i,j,k;
	
	float *d_A, *d_B, *d_C;	
	float *h_A, *h_B, *h_C;

	size_t size = N*N*sizeof(float);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Allocate Device memory
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	//Allocate Host memory
	hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
	hipHostMalloc((void **)&h_B, size, hipHostMallocDefault);	
	hipHostMalloc((void **)&h_C, size, hipHostMallocDefault);

	
	//Initialize matrices on the host
	for(i=0;i<N;i++){
	    for(j=0;j<N;j++){
		h_A[i*N+j]=i;
		h_B[i*N+j]=i+1;
	    }
	}


	//Allocate A and B to the Device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


	//Invoke kernel
	dim3 blockPerGrid(N/TILE_SIZE,N/TILE_SIZE);
	dim3 threadPerBlock(TILE_SIZE,TILE_SIZE);

	hipEventRecord(start);	
	MatrixMultiply<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, d_C,N);
	hipEventRecord(stop);	


	//Read C from device
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Elapsed time (ms): %f\n", milliseconds);
	
	//Calculate the MM result with normal CPU implementation and compare the results with the GPU

	float * test_C;
	test_C = (float *)malloc(size);	
	for (i=0; i<N; i++){
		for (j=0;j<N;j++){
			float sum = 0;
			for (k=0;k<N;k++){
				float a = h_A[i*N+k];
				float b = h_B[k*N+j];
				sum += a*b;
			}
			test_C[i*N+j]= sum;
		}
	}
	int compare = 0;
	for(i=0;i<N;i++){
		for(j=0;j<N;j++){
			if (test_C[i*N+j]==h_C[i*N+j]){
				compare++;
			}else{
				compare+=0;
			}
		}
	}
	if(compare == N*N){
		printf("Success!\n");
	}else{
		printf("Error!\n");	
	}

	/*=============================Finish Test=================================*/

	free(test_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(h_A);
	hipFree(h_B);
	hipFree(h_C);
	hipDeviceReset();
	return EXIT_SUCCESS;
}
