
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#define N 22

__global__ void MatrixMultiply(float *d_A, float *d_B, float *d_C)
{
	
	int ty = threadIdx.y;
	int tx = threadIdx.x;
	
	float Cvalue = 0.0;	
	int i;


	for(i=0; i<N; i++){
		float d_Aelment = d_A[ty*N + i];
		float d_Belment = d_B[i*N + tx];
		Cvalue += d_Aelment * d_Belment;		
	}
	d_C[ty*N + tx]= Cvalue;
}

int main(){

	int i,j,k;
	
	float *d_A, *d_B, *d_C;	
	float *h_A, *h_B, *h_C;

	size_t size = N*N*sizeof(float);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Allocate Device memory
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	//Allocate Host memory
	hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
	hipHostMalloc((void **)&h_B, size, hipHostMallocDefault);	
	hipHostMalloc((void **)&h_C, size, hipHostMallocDefault);

	
	//Initialize matrices on the host
	for(i=0;i<N;i++){
	    for(j=0;j<N;j++){
		h_A[i*N+j]=i;
		h_B[i*N+j]=i+1;
	    }
	}


	//Allocate A and B to the Device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


	//Invoke kernel
	dim3 blockPerGrid(1,1);
	dim3 threadPerBlock(N,N);

	hipEventRecord(start);	
	MatrixMultiply<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop);	


	//Read C from device
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Elapsed time (ms): %f\n", milliseconds);
	
	//Calculate the MM result with normal CPU implementation and compare the results with the GPU

	float * test_C;
	test_C = (float *)malloc(size);	
	for (i=0; i<N; i++){
		for (j=0;j<N;j++){
			float sum = 0;
			for (k=0;k<N;k++){
				float a = h_A[i*N+k];
				float b = h_B[k*N+j];
				sum += a*b;
			}
			test_C[i*N+j]= sum;
		}
	}
	int compare = 0;
	for(i=0;i<N;i++){
		for(j=0;j<N;j++){
			if (test_C[i*N+j]==h_C[i*N+j]){
				compare++;
			}else{
				compare+=0;
			}
		}
	}
	if(compare == N*N){
		printf("Success!\n");
	}else{
		printf("Error!\n");	
	}

	/*=============================Finish Test=================================*/

	free(test_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(h_A);
	hipFree(h_B);
	hipFree(h_C);
	hipDeviceReset();
	return EXIT_SUCCESS;
}
