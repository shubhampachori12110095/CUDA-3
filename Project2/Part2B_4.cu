
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 1024

__global__ void MatrixMultiply(float *d_A, float *d_B, float *d_C, int NB, int NT)
{
	
	int numrow = NB*(blockIdx.y*NT + threadIdx.y);
	int numcol = NB*(blockIdx.x*NT+threadIdx.x);
	
	for(int j=0;j<NB;j++){
		for (int i=0;i<NB;i++){
		float cvalue=0.0;
			for (int k=0;k<N;k+=4){

		cvalue+= d_A[(j+numrow)*N +k] * d_B[k*N + (numcol+i)];
		cvalue+= d_A[(j+numrow)*N +k+1] * d_B[(k+1)*N + (numcol+i)];		
		cvalue+= d_A[(j+numrow)*N +k+2] * d_B[(k+2)*N + (numcol+i)];
		cvalue+= d_A[(j+numrow)*N +k+3] * d_B[(k+3)*N + (numcol+i)];
			}
		d_C[(j+numrow)*N + numcol+i] = cvalue;}}
}

int main(){

	int i,j,k;
	int NB,NT;
	float *d_A, *d_B, *d_C;	
	float *h_A, *h_B, *h_C;

	size_t size = N*N*sizeof(float);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Allocate Device memory
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	//Allocate Host memory
	hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
	hipHostMalloc((void **)&h_B, size, hipHostMallocDefault);	
	hipHostMalloc((void **)&h_C, size, hipHostMallocDefault);

	
	//Initialize matrices on the host
	for(i=0;i<N;i++){
	    for(j=0;j<N;j++){
		h_A[i*N+j]=i;
		h_B[i*N+j]=i+1;
	    }
	}


	//Allocate A and B to the Device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	NB = 4;
	NT = 64;
	//Invoke kernel
	dim3 blockPerGrid(N/(NT*NB),N/(NT*NB));
	dim3 threadPerBlock(NT,NT);
	
	hipEventRecord(start);	
	for (int s=0;s<5;s++){
	MatrixMultiply<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, d_C,NB,NT);
	}
	hipEventRecord(stop);	
	

	//Read C from device
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("NB %d NK %d NT %d Elapsed time (ms): %f\n",NB,N/(NT*NB),NT, milliseconds/5);

	//Calculate the MM result with normal CPU implementation and compare the results with the GPU

	float * test_C;
	test_C = (float *)malloc(size);	
	for (i=0; i<N; i++){
		for (j=0;j<N;j++){
			float sum = 0;
			for (k=0;k<N;k++){
				float a = h_A[i*N+k];
				float b = h_B[k*N+j];
				sum += a*b;
			}
			test_C[i*N+j]= sum;
		}
	}
	int compare = 0;
	for(i=0;i<N;i++){
		for(j=0;j<N;j++){
			if (test_C[i*N+j]==h_C[i*N+j]){
				compare++;
			}else{
				compare+=0;
			}
		}
	}
	if(compare == N*N){
		printf("Success!\n");
	}else{
		printf("Error!\n");	
	}

	/*=============================Finish Test=================================*/

	free(test_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(h_A);
	hipFree(h_B);
	hipFree(h_C);
	hipDeviceReset();
	return EXIT_SUCCESS;
}
