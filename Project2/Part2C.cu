
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 1024

__global__ void MatrixMultiply(float *d_A, float *d_B, float *d_C, int NB, int NT)
{
	__shared__ float Mds[64*64];
	__shared__ float Nds[64*64];
	int tx =threadIdx.x;
	int ty =threadIdx.y;
	int numrow = NB*(blockIdx.y*NT + threadIdx.y);
        int numcol = NB*(blockIdx.x*NT+threadIdx.x);


	for (int j=0;j<NB;j++){
		for (int i=0;i<NB;i++){		
        	float cvalue=0.0;
       		for (int m=0; m < N/(NT*NB); ++m){
	Mds[(ty+j)*NT*NB+tx+i] = d_A[(j+numrow)*N + (m*NT*NB+tx+i)];
	Nds[(ty+j)*NT*NB+tx+i] = d_B[numcol+i + (m*NT*NB + ty+j)*N];
	__syncthreads();
                        for (int k=0;k<NB*NT;k+=4){
                cvalue+= Mds[(ty+j)*NT*NB+k] * Nds[k*NT*NB+tx+i];
		cvalue+= Mds[(ty+j)*NT*NB+k+1] * Nds[(k+1)*NT*NB+tx+i];

		cvalue+= Mds[(ty+j)*NT*NB+k+2] * Nds[(k+2)*NT*NB+tx+i];

		cvalue+= Mds[(ty+j)*NT*NB+k+3] * Nds[(k+3)*NT*NB+tx+i];

                        }
                d_C[(j+numrow)*N+numcol+i] = cvalue;
}}}}


int main(){

	int i,j,k;
	int NB =1;
	int NT= 64;
	float *d_A, *d_B, *d_C;	
	float *h_A, *h_B, *h_C;

	size_t size = N*N*sizeof(float);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Allocate Device memory
	hipMalloc((void **)&d_A, size);
	hipMalloc((void **)&d_B, size);
	hipMalloc((void **)&d_C, size);

	//Allocate Host memory
	hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
	hipHostMalloc((void **)&h_B, size, hipHostMallocDefault);	
	hipHostMalloc((void **)&h_C, size, hipHostMallocDefault);

	
	//Initialize matrices on the host
	for(i=0;i<N;i++){
	    for(j=0;j<N;j++){
		h_A[i*N+j]=i;
		h_B[i*N+j]=i+1;
	    }
	}


	//Allocate A and B to the Device
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	/*for(NT=32;NT<N;NT=2*NT){*/
	/*for(NB=2;NB<(N/NT);NB=NB*2){*/

	//Invoke kernel
	dim3 blockPerGrid(N/(NT*NB),N/(NT*NB));
	dim3 threadPerBlock(NT,NT);

	hipEventRecord(start);	
	MatrixMultiply<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, d_C,NB,NT);
	hipEventRecord(stop);	


	//Read C from device
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("NB %d NK %d NT %d Elapsed time (ms): %f\n",NB,N/(NT*NB),NT, milliseconds);

	//Calculate the MM result with normal CPU implementation and compare the results with the GPU

	float * test_C;
	test_C = (float *)malloc(size);	
	for (i=0; i<N; i++){
		for (j=0;j<N;j++){
			float sum = 0;
			for (k=0;k<N;k++){
				float a = h_A[i*N+k];
				float b = h_B[k*N+j];
				sum += a*b;
			}
			test_C[i*N+j]= sum;
		}
	}
	int compare = 0;
	for(i=0;i<N;i++){
		for(j=0;j<N;j++){
			if (test_C[i*N+j]==h_C[i*N+j]){
				compare++;
			}else{
				compare+=0;
			}
		}
	}
	if(compare == N*N){
		printf("Success!\n");
	}else{
		printf("Error!\n");	
	}

	/*=============================Finish Test=================================*/

	free(test_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(h_A);
	hipFree(h_B);
	hipFree(h_C);
	hipDeviceReset();
	return EXIT_SUCCESS;
}
